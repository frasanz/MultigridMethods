
/*
 * This is the jacobi relaxation method in gpu 
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SIZE 2048
#define BLOCK_SIZE 32
#define NITER 1000

float ratio(float*u, float ant, int iter){
	float tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[j*SIZE+i]>tmp)
				tmp=u[j*SIZE+i];
		}
	}
	printf(" iter=%d ratio=%f ant=%f max=%f\n",iter,tmp/ant,ant,tmp);
	return tmp;
}

__global__ void jacobi(float *d_u_new, float *d_u, float *d_f, float h2){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	d_u_new[i*SIZE+j]=0.25*(
			h2*d_f[i    *SIZE+j     ]+
			d_u[(i-1)*SIZE+j     ]+
			d_u[(i+1)*SIZE+j     ]+
			d_u[i    *SIZE+j-1   ]+
			d_u[i    *SIZE+j+1   ]);


}

int main(){
	float * h_u, *h_f;
	float * d_u, *d_u_new, *d_f;
	float * tmp;
	float ant = 1.0;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;

	/* Host memory malloc */
	size = SIZE*SIZE*sizeof(float);
	printf("We need %dmb of memory\n",3*size/1024/1024);
	h_u = (float*)malloc(size);
	h_f = (float*)malloc(size);

	/* memory for the gpu */
	hipMalloc(&d_u, size);
	hipMalloc(&d_u_new, size);
	hipMalloc(&d_f, size);

	/* Initialization */
	for(i=0;i<SIZE; i++){
		for(j=0; j<SIZE; j++){
			h_f[i*SIZE+j]=0.0;
			h_u[i*SIZE+j]=rand();
		}
	}

	/* Bounds */
	for(i=0;i<SIZE;i++){
		h_u[i]=0.0;
		h_u[i*SIZE]=0.0;
		h_u[i*SIZE+SIZE-1]=0.0;
		h_u[SIZE*(SIZE-1)+i]=0.0;
	}
	/* Copy from host to device */
	hipMemcpy(d_f,h_f,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u,h_u,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u_new,h_u,size,hipMemcpyHostToDevice);

	/* Grid dimension */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);
	float h2=h*h;

	/* Call NITER times to the jacobi method */
	for(i=0;i<NITER;i++)
	{
		jacobi<<<dimGrid,dimBlock>>>(d_u_new,d_u,d_f,h2);
		hipDeviceSynchronize;
		if(i%100==0){
			hipMemcpy(h_u, d_u_new, size, hipMemcpyDeviceToHost);
			ant=ratio(h_u,ant,i);
		}
		tmp=d_u_new;
		d_u_new=d_u;
		d_u=tmp;

	}

	/* free memory */
	free(h_u);
	free(h_f);
	hipFree(d_u_new);
	hipFree(d_u);
	hipFree(d_f);
}
