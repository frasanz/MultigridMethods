
#include <hip/hip_runtime.h>
#define IDT(i,j) (i)*((i)+1)/2+(j)

typedef struct{
	double *v;
	int dim;
	int size;
} Grid;

__global__ void cero(Grid m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=m.dim-1 && j<=i){
		if(j==0 || i==m.dim-1 || i==j)
			m.v[IDT(i,j)]=0.0;
		else
			m.v[IDT(i,j)]=0.0;
	}
}

__global__ void uno(Grid m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=m.dim-1 && j<=i){
		if(j==0 || i==m.dim-1 || i==j)
			m.v[IDT(i,j)]=0.0;
		else
			m.v[IDT(i,j)]=0.0;
	}
}

__global__ void inicializa_f(Grid m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=m.dim-1 && j<=i)
	{
		if(j==0 || i==m.dim-1 || i==j)
			m.v[IDT(i,j)]=0.0;
		if(j>0 && j<i && i<m.dim-1)
			m.v[IDT(i,j)]=4.0;
	}
}

__global__ void random(Grid m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=m.dim-1 && j<=i)
	{
		if(j==0 || i==m.dim-1 || i==j)
			m.v[IDT(i,j)]=0.0;
		if(j>0 && j<i && i<m.dim-1)
			m.v[IDT(i,j)]=i^2+j^2;
	}
}

__global__ void suaviza_r(Grid u, Grid f, double * op)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=u.dim-1 && j<=i)
	{

		if(j>0 && j<i && i<u.dim-1)
			if((i+j)%3==0)
			{
				u.v[IDT(i,j)]=(f.v[IDT(i,j)]-op[0]*u.v[IDT(i-1,j-1)]
						-op[1]*u.v[IDT(i-1,j  )]
						-op[2]*u.v[IDT(i-1,j+1)]
						-op[3]*u.v[IDT(i  ,j-1)]
						-op[5]*u.v[IDT(i  ,j+1)]
						-op[6]*u.v[IDT(i+1,j-1)]
						-op[7]*u.v[IDT(i+1,j  )]
						-op[8]*u.v[IDT(i+1,j+1)])/op[4];
			}
	}
}

__global__ void suaviza_g(Grid u, Grid f, double * op)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=u.dim-1 && j<=i)
	{

		if(j>0 && j<i && i<u.dim-1)
			if((i+j)%3==1)
			{
				u.v[IDT(i,j)]=(f.v[IDT(i,j)]-op[0]*u.v[IDT(i-1,j-1)]
						-op[1]*u.v[IDT(i-1,j  )]
						-op[2]*u.v[IDT(i-1,j+1)]
						-op[3]*u.v[IDT(i  ,j-1)]
						-op[5]*u.v[IDT(i  ,j+1)]
						-op[6]*u.v[IDT(i+1,j-1)]
						-op[7]*u.v[IDT(i+1,j  )]
						-op[8]*u.v[IDT(i+1,j+1)])/op[4];
			}
	}
}


__global__ void suaviza_b(Grid u, Grid f, double * op)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=u.dim-1 && j<=i)
	{

		if(j>0 && j<i && i<u.dim-1)
			if((i+j)%3==2)
			{
				u.v[IDT(i,j)]=(f.v[IDT(i,j)]-op[0]*u.v[IDT(i-1,j-1)]
						-op[1]*u.v[IDT(i-1,j  )]
						-op[2]*u.v[IDT(i-1,j+1)]
						-op[3]*u.v[IDT(i  ,j-1)]
						-op[5]*u.v[IDT(i  ,j+1)]
						-op[6]*u.v[IDT(i+1,j-1)]
						-op[7]*u.v[IDT(i+1,j  )]
						-op[8]*u.v[IDT(i+1,j+1)])/op[4];
			}
	}
}

__global__ void defecto(Grid u, Grid f, Grid d, double * op)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if(i<=u.dim-1 && j<=i)
	{
		if(j>0 && j<i && i<u.dim-1 && i>0) /* puntos interiores */
		{
			d.v[IDT(i,j)]=f.v[IDT(i,j)]
				-op[1]*u.v[IDT(i-1,j)]
				-op[2]*u.v[IDT(i-1,j+1)]
				-op[3]*u.v[IDT(i,j-1)]
				-op[4]*u.v[IDT(i,j)]
				-op[5]*u.v[IDT(i,j+1)]
				-op[6]*u.v[IDT(i+1,j-1)]
				-op[7]*u.v[IDT(i+1,j)]
				-op[8]*u.v[IDT(i+1,j+1)];
		}
	}
}

__global__ void restringe(Grid sup, Grid inf)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(j>0 && j<i && i<inf.dim-1 && i>0) /* puntos interiores de la malla inferior*/
	{
		inf.v[IDT(i,j)]=(sup.v[IDT(2*i,2*j)]+0.5*(
					sup.v[IDT(2*i-1,2*j-1)]
					+sup.v[IDT(2*i-1,2*j  )]
					+sup.v[IDT(2*i  ,2*j-1)]
					+sup.v[IDT(2*i  ,2*j+1)]
					+sup.v[IDT(2*i+1,2*j  )]
					+sup.v[IDT(2*i+1,2*j+1)]))/4; 

	}
}
__global__ void interpola(Grid inf, Grid sup)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=inf.dim-1 && j<=i)
	{

		if(j>0 && j<i && i<inf.dim-1 && i>0) /* puntos interiores de la malla inferior*/
		{
			sup.v[IDT(2*i  ,2*j  )] =  inf.v[IDT(i,j)];
			sup.v[IDT(2*i-1,2*j-1)] = (inf.v[IDT(i,j)]+inf.v[IDT(i-1,j-1)])/2;
			sup.v[IDT(2*i-1,2*j  )] = (inf.v[IDT(i,j)]+inf.v[IDT(i-1,j  )])/2;
			sup.v[IDT(2*i  ,2*j+1)] = (inf.v[IDT(i,j)]+inf.v[IDT(i  ,j+1)])/2;
			sup.v[IDT(2*i+1,2*j+1)] = (inf.v[IDT(i,j)]+inf.v[IDT(i+1,j+1)])/2;
			sup.v[IDT(2*i+1,2*j  )] = (inf.v[IDT(i,j)]+inf.v[IDT(i+1,j  )])/2;
			sup.v[IDT(2*i  ,2*j-1)] = (inf.v[IDT(i,j)]+inf.v[IDT(i-1,j-1)])/2;
		}
	}
}
__global__ void suma(Grid u, Grid v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.dim && j<=i)
	{
		u.v[IDT(i,j)]=u.v[IDT(i,j)]+v.v[IDT(i,j)];
	}
}





