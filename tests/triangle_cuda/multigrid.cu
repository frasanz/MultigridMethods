#include "hip/hip_runtime.h"
#include "multigrid_kernel.cu"
#include <stdio.h>
#define N_MALLAS 12
#define BLOCK_SIZE 16

void g_imprime(Grid g);
void multigrid(Grid *u, Grid *f, Grid *v, Grid *d, int nivel, double *max, int *iter);

void imprime_malla(double *f, int dim, const char * nombre){
	FILE *fil;
	fil = fopen(nombre,"w");
	int i,j;
	double h=1.0/(dim-1);
	for( i=0; i<dim; i++) {
		for( j=0; j<=i; j++) {
			fprintf(fil,"%f %f %f\n", 1.0*j*h, 1.0-1.0*i*h, f[IDT(i,j)]);
		}
		fprintf(fil,"\n");
	}
	fclose(fil);
}

int main(){
	int i;
	int dim;
	int size;
	double max=100;
	double max_ant;
	int sizetotal=0;
	
	/* Definition of the grid */
	Grid u[N_MALLAS];
	Grid f[N_MALLAS];
	Grid v[N_MALLAS];
	Grid d[N_MALLAS];

	/* Memory alloc */
	for(i=2; i<N_MALLAS; i++){
		dim=pow(2,i)+1; //Dim is the number of elements in the diag
		size = ((dim-1)*(dim-1)+3*(dim-1))/2+2;
		u[i].dim=dim;
		f[i].dim=dim;
		v[i].dim=dim;
		d[i].dim=dim;
		u[i].size=size;
		f[i].size=size;
		v[i].size=size;
		d[i].size=size;

		hipMalloc(&u[i].v,size*sizeof(double));
		hipMalloc(&f[i].v,size*sizeof(double));
		hipMalloc(&v[i].v,size*sizeof(double));
		hipMalloc(&d[i].v,size*sizeof(double));
		sizetotal = sizetotal+4*size;
	}


	/* To CALL CUDA */
	int m = N_MALLAS -1;
	dim = (int) pow(2,m)+1;
	dim3 dimBlock (BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x, (dim+BLOCK_SIZE-1)/dimBlock.y);
	printf("%d %d %d\n",dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y);
	printf("We need about %d Mb\n", sizetotal*sizeof(double)/1024/1024);

	/* Starting the grid of f */
	cero<<<dimGrid, dimBlock>>>(f[m]);

	/* Initialize u with random values */
	random <<<dimGrid, dimBlock>>>(u[m]);

	/* Main loop */
	int iter=0;
	for(i=0;i<20;i++){
		max_ant = max;
		max =0.0;
		multigrid(&u[0], &f[0], &v[0], &d[0], m, &max, &iter);
		printf("Iteration %d nd=%d ratio=%d\n", i, max, max/max_ant);
		iter++;
	}

	/* Free memory */
	for(i=0; i< N_MALLAS; i++){
		hipFree(&u[i].v);
		hipFree(&f[i].v);
		hipFree(&v[i].v);
		hipFree(&d[i].v);
	}
	return 0;
}

/* This function prints a grid located in the GPU */
void g_print(Grid g, const char *name){
	double *dg;
	FILE *file;
	file = fopen(name,"w");
	double h=1.0/(g.dim-1);
	int i,j;
	size_t size=((g.dim-1)*(g.dim-1)+3*(g.dim-1))/2+1;
	dg = (double*)malloc(size*sizeof(double));
	
	hipMemcpy(dg,g.v, size*sizeof(double), hipMemcpyDeviceToHost);

	for(i=0;i<g.dim;i++){
		for(j=0;j<=i;j++){
			fprintf(file,"%f %f %f\n",1.0*j*h,1.0-1.0*i*h,dg[IDT(i,j)]);
		}
		fprintf(file,"\n");
	}
	fclose(file);
	free(dg);
}

void multigrid(Grid *u, Grid *f, Grid *v, Grid *d, int m, double *max, int *iter){
	int dim;
	int dim_;
	int i,j;
	double * hf;
	double * hu;


	/* Definition of h^2 */
	double h2=pow(u[m].dim-1,2);

	/* Definition of an operador (copied from another site) */
	double operador[9]={0.0,-1.0*h2,0.0,-1.0*h2,4.0*h2,-1.0*h2,0.0,-1.0*h2,0.0};
	double * a_op;
	hipMalloc(&a_op,9*sizeof(double));
	hipMemcpy(a_op,&operador[0],9*sizeof(double),hipMemcpyHostToDevice);


	if(m==2){ /* In this case, we've to solve */
		dim = (int)pow(2,m)+1;
		size_t size=((f[m].dim-1)*(f[m].dim-1)+3*(f[m].dim-1))/2+1;
		hf=(double*)malloc(size*sizeof(double));
		hipMemcpy(hf,f[m].v,size*sizeof(double),hipMemcpyDeviceToHost);
		hu=(double*)malloc(size*sizeof(double));

		/* Construimos el sistema a resolver */
		double A[3][3];
		A[0][0]=operador[4];
		A[0][1]=operador[7];
		A[0][2]=operador[8];
		A[1][0]=operador[2];
		A[1][1]=operador[4];
		A[1][2]=operador[5];
		A[2][0]=operador[0];
		A[2][1]=operador[3];
		A[2][2]=operador[4];

		double B[3];
		B[0]=hf[IDT(2,1)];
		B[1]=hf[IDT(3,1)];
		B[2]=hf[IDT(3,2)];

		/* Hacemos eliminación gausiana */
		A[1][1]=A[1][1]-A[0][1]*A[1][0]/A[0][0];
		A[1][2]=A[1][2]-A[0][2]*A[1][0]/A[0][0];
		B[1]=B[1]-B[0]*A[1][0]/A[0][0];
		A[2][1]=A[2][1]-A[0][1]*A[2][0]/A[0][0];
		A[2][2]=A[2][2]-A[0][2]*A[2][0]/A[0][0];
		B[2]=B[2]-B[0]*A[2][0]/A[0][0];
		A[2][2]=A[2][2]-A[1][2]*A[2][1]/A[1][1];
		B[2]=B[2]-B[1]*A[2][1]/A[1][1];

		/* Resolvemos */
		hu[IDT(3,2)]=B[2]/A[2][2];
		hu[IDT(3,1)]=(B[1]-A[1][2]*hu[IDT(3,2)])/A[1][1];
		hu[IDT(2,1)]=(B[0]-A[0][2]*hu[IDT(3,2)]-A[0][1]*hu[IDT(3,1)])/A[0][0];

		/* Subimos la solución a la GPU */
		hipMemcpy(u[m].v,hu,size*sizeof(double),hipMemcpyHostToDevice);
		free(hf);
	} else {
		/* To call CUDA */
		dim =(int)pow(2,m)+1;
		dim_=(int)pow(2,m-1)+1;
		dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
		dim3 dimGrid ((dim +BLOCK_SIZE-1)/dimBlock.x,(dim +BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_((dim_+BLOCK_SIZE-1)/dimBlock.x,(dim_+BLOCK_SIZE-1)/dimBlock.y);

		/* Set 0 in the appropiated grids */
		cero<<<dimGrid ,dimBlock>>>(v[m]);
		cero<<<dimGrid ,dimBlock>>>(d[m]);
		cero<<<dimGrid_,dimBlock>>>(u[m-1]);
		cero<<<dimGrid_,dimBlock>>>(f[m-1]);

		/* smooth three colors */
		suaviza_r<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_g<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_b<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);

		/* smooth three colors */
		suaviza_r<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_g<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_b<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);

		/* Compute the defect */
		defecto<<<dimGrid, dimBlock>>>(u[m],f[m],d[m],a_op);

		/* Restrict the defect */
		restringe<<<dimGrid_, dimBlock>>>(d[m], f[m-1]);


		/* Recall to multigrid */
		for(i=0; i<2; i++){
			multigrid(&u[0],&f[0],&v[0],&d[0],m-1,max,iter);
		}

		/* Interpolate from u[m-1] to v[m] */
		interpola<<<dimGrid_, dimBlock>>>(u[m-1],v[m]);

		/* Sum */
		suma<<<dimGrid_, dimBlock>>>(u[m],v[m]);

		/* smooth three colors */
		suaviza_r<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_g<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_b<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);

		/* smooth three colors */
		suaviza_r<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_g<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);
		suaviza_b<<<dimGrid, dimBlock>>>(u[m],f[m],a_op);

		/* In the uppper grid, check defect */
		if(m==N_MALLAS-1){
			char nombre[256];
			sprintf(nombre,"defecto_%d",iter[0]);
			defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m],a_op);
			double *def;
			size_t size=((f[m].dim-1)*(f[m].dim-1)+3*(f[m].dim-1))/2+1;
			def=(double*)malloc(size*sizeof(double));
			hipMemcpy(def,d[m].v,size*sizeof(double), hipMemcpyDeviceToHost);
			for(i=0;i<size;i++)
			{
				if(max[0]<fabs(def[i]))
					max[0]=fabs(def[i]);
			}
			free(def);
		}
	}
	hipFree(a_op);
}
