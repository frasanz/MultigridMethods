#include "hip/hip_runtime.h"
// system includes
#include <stdio.h>
#include <assert.h>


// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions
#include "helper_functions.h"

template <int BLOCK_SIZE> __global__ void matrixMulCUDA(float *C, float *A, float *B, int wA, int wB){

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first submatrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last submatrix of A processed by the block
	int aEnd = aBegin + wA -1 ;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first submatrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the submatrices of B
	int bStep = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
	float cSub = 0;

		// Loop over all the submatrices of A and B required to compute the block submatrix
		for(int a = aBegin, b = bBegin; a <=aEnd; a += aStep , b+= bStep){
			// Declaration of the shared memory array As used to store the submatrix of A
			__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

			// Declaration of the shared memory array Bs used to store the submatrix of B
			__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

			// Load the matrices from device memory to shared memory, each thread loads 
			// one element of each matrix
			As[ty][tx] = A[a + wA * ty + tx];
			Bs[ty][tx] = B[b + wB * ty + tx];

			// Synchronize to make sure the matrices are loaded
			__syncthreads();

			// Multiply the two matrices together;
			// each thread computes one element
			// of the block submatrix
#pragma unroll

			for (int k=0; k < BLOCK_SIZE ; ++k){
				cSub += As[ty][k] * Bs[k][tx];
		}
		// Synchronize to make sure that the preceding caomputation is done begore loading
		// two new sub-matrices of A and B in the next iteration
		__syncthreads();
		}

		int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
		C[c + wB * ty + tx ]=cSub;
}

void constantInit(float * data, int size, float val){
	for (int i=0; i< size ; i++){
		data[i] = val;
	}
}

/**
 * Run a simple test of matrix multiplication using CUDA 
 */

int matrixMultiply(int argc, char ** argv, int block_size, dim3 &dimsA, dim3 &dimsB){
	// Allocate host memory for matrices A and B
	unsigned int size_A     = dimsA.x * dimsA.y;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float *h_A = (float *)malloc(mem_size_A);
	unsigned int size_B = dimsB.x * dimsB.y;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float *h_B = (float *)malloc(mem_size_B);

	// Initialize host memory
	const float valB = 0.01f;
	constantInit(h_A, size_A, 1.0f);
	constantInit(h_B, size_B, valB);

	// Allocate device memory
	float *d_A, *d_B, *d_C;

	// Allocate host matrix C
	dim3 dimsC(dimsB.x, dimsA.y, 1);
	unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
	float *h_C = (float *) malloc(mem_size_C);

	if(h_C == NULL){
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	hipError_t error;

	error = hipMalloc((void **) &d_A, mem_size_A);
	if(error != hipSuccess){
		printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **) &d_B, mem_size_B);
	if(error != hipSuccess){
		printf("cudaMallco d_B returned error code %d, line (%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **) &d_C, mem_size_C);
	if(error != hipSuccess){
		printf("hipMalloc d_C returned error code %d, line (%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	if(error != hipSuccess){
		printf("hipMemcpy (d_B, h_B) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Setup execution parameters
	dim3 threads(block_size, block_size);
	dim3 grids(dimsB.x / threads.x, dimsA.y / threads.y);

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Performs warmup operation using matrixMul CUDA kernel
	if(block_size == 16){
		matrixMulCUDA<16><<<grids, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	} else {
		matrixMulCUDA<32><<<grids, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	}
	printf("done\n");

	hipDeviceSynchronize();

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 300;
	for (int j=0; j< nIter; j++){
		if(block_size ==16){
			matrixMulCUDA<16><<<grids, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
		} else {
			matrixMulCUDA<32><<<grids, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
		}
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess){
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
			hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	float  msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n",
			hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the perfomance
	float msecPerMatrixMul   = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf( "Perfomance = %.2f GFlops/s, Time       = %.3f msec, Size       = %.0f Ops, WorkGoupSiz= %u threads/block\n",
		 	gigaFlops,
			msecPerMatrixMul,
			flopsPerMatrixMul,
			threads.x * threads.y);

	// Copy results from device to host
	error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
	if(error != hipSuccess){
		printf("hipMemcpy (h_C, d_C) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	printf("Checking computed result for correctness: ");
	bool correct = true;

	// test relative error by the formula
	// |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|> <eps
	double eps = 1.e-6; // machine zero
	for(int i = 0; i < (int) (dimsC.x * dimsC.y); i++){
		double abs_err = fabs(h_C[i] - (dimsA.x * valB));
		double dot_length = dimsA.x;
		double abs_val = fabs(h_C[i]);
		double rel_err = abs_err/abs_val/dot_length;
		if ( rel_err > eps){
			printf("Error! Matrix[%05d]=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
			correct = false;
		}
	}

	printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	printf("\nNote: For peak perfomance, please refer to the matrixMulCUBLAS example.\n");

	hipDeviceReset();
	if(correct)
		return EXIT_SUCCESS;
	else
		return EXIT_FAILURE;
}

/* *
 * Program main
 */

int main(int argc, char **argv){

	printf("[Matrix Multiply Using CUDA] - Starting...\n");

	if(checkCmdLineFlag(argc, (const char **)argv, "help") ||
		 checkCmdLineFlag(argc, (const char **)argc, "?")){
		printf("Usage -device=n (n>=0 for deviceID)\n");
		printf("\t-wA=WidthA -hA=HeightA (Width * Height of Matrix A)\n");
		printf("\t-wB=WidthB -hB=HeightB (Width * Height of Matrix B)\n");
		printf("Note: Outer matrix dimensions of A & B matrices must be equal.\n");

		exit(EXIT_SUCCESS);
	}

	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	int devID = 0;

	if(checkCmdLineFlag(argc, (const char **)argv, "device")){
		devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
		hipSetDevice(devID);
	}

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if(error != hipSuccess){
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}
	error = hipGetDeviceProperties(&deviceProp, devID);

	if(deviceProp.computeMode == hipComputeModeProhibited){
		fprintf(stderr, "error,: device is running in <Compute Mode Prohibited>, no threads ca use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}
	if(error != hipSuccess){
		printf("cudaGetFeviceProperties returned error code %d, line (%d)\n", error, __LINE__);
	} else {
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, 
			deviceProp.major, deviceProp.minor);
	}

	// Use a larger block size for Fermi and above
	int block_size = (deviceProp.major < 2) ? 16 : 32;

	dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
	dim3 dimsB(5*4*block_size, 5*2*block_size,1);	

	 // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}
