#include "hip/hip_runtime.h"
//This is an example on vectorAdd just to remember how CUDA works

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if(i<numElements){
    C[i] = A[i]+B[i];
  }
}

int main(void){
  hipError_t err = hipSuccess;

  int numElements=50000;
  size_t size = numElements*sizeof(float);
  printf("[Vector addition of %d elements\n", numElements);
  printf("[We will need %d bytes]\n", size);

  // Allocate in host for input A/B outputC
  float *h_A = (float*)malloc(size);
  float *h_B = (float*)malloc(size);
  float *h_C = (float*)malloc(size);

  //Is it ok?
  if(h_A == NULL || h_B == NULL || h_C == NULL){
    fprintf(stderr, "Failure in allocation\n");
    exit(EXIT_FAILURE);
  }

  // Initializing in host
  for(int i=0; i< numElements; i++){
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
  }

  // Allocate in the device input vector *A
  float *d_A = NULL;
  err = hipMalloc((void**)&d_A, size);
  if(err!=hipSuccess){
     fprintf(stderr, "Failed(error code %s)!\n", hipGetErrorString(err));
    printf("error1\n");
    exit(EXIT_FAILURE);
  }
  float *d_B = NULL;
  err = hipMalloc((void**)&d_B, size);
  if(err!=hipSuccess){
    printf("error2\n");

    exit(EXIT_FAILURE);
  }

  float *d_C = NULL;
  err = hipMalloc((void**)&d_C, size);
  if(err!=hipSuccess){
    exit(EXIT_FAILURE);
  }

  // Copying
  err = hipMemcpy(d_A,h_A,size, hipMemcpyHostToDevice);
  if(err!=hipSuccess){
    printf("error3\n");

    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B,h_B,size, hipMemcpyHostToDevice);
  if(err!=hipSuccess){
     fprintf(stderr, "(error code %s)!\n", hipGetErrorString(err));

    printf("error7\nn");

    exit(EXIT_FAILURE);
  }

  // launch the Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock -1) / threadsPerBlock;
  printf("Launching %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,d_C, numElements);
  err = hipGetLastError();
  if (err!=hipSuccess){
    printf("error4\n");

    exit(EXIT_FAILURE);
  }

  // COpyback the solution
  err = hipMemcpy(h_C,d_C,size, hipMemcpyDeviceToHost);
  if(err !=hipSuccess){
    printf("error5\n");
    fprintf(stderr, "(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Verify solution
  for(int i=0;i<numElements; i++){
    if(fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5 ){
    printf("error6\n");

      exit(EXIT_FAILURE);
    }
  }
  printf("Test PASSED\n");

 hipFree(d_A);
 hipFree(d_B);
 hipFree(d_C);
 free(h_A);
 free(h_C);
 free(h_B);
}
