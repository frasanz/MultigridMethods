#include "hip/hip_runtime.h"
#include <stdio.h>
#include "multigrid_kernel.cu"

#define N_MALLAS 12
#define BLOCK_SIZE 16

void gpu_imprime(Grid g, const char *);
void gpu_muestra(Grid g, const char *);
void multigrid(Grid *u, 
               Grid *f, 
							 Grid *v, 
							 Grid *d, 
							 int m,
							 double *max,
							 double *def,
							 double *host_def); 


int main(){
	int i;
	int dim;
	int size;
	int sizetotal=0;
	double max=1.0;
	double max_ant;

	/* Definition of the Grids */
	Grid u[N_MALLAS];
	Grid f[N_MALLAS];
	Grid v[N_MALLAS];
	Grid d[N_MALLAS];

	/* Double to compute max(defect) of each file */
	double * gpu_def;    //In GPU
	double * host_def;   //In Host

	/* malloc */
	for(i=0; i<N_MALLAS; i++){
		dim  =  (int)pow(2,i+1)+1;
		size =  dim*dim;
		u[i].d = dim;
		f[i].d = dim;
		v[i].d = dim;
		d[i].d = dim;

		u[i].size = size;
		f[i].size = size;
		v[i].size = size;
		d[i].size = size;

		hipMalloc(&u[i].v, size*sizeof(double));
		hipMalloc(&v[i].v, size*sizeof(double));
		hipMalloc(&d[i].v, size*sizeof(double));
		hipMalloc(&f[i].v, size*sizeof(double));
		sizetotal +=4*size;
	}
	int m = N_MALLAS - 1 ;
	dim = (int)pow(2,m+1)+1;
	size = dim*dim;
	hipMalloc(&gpu_def, size*sizeof(double));
	sizetotal+=size;
	host_def=(double*)malloc(size*sizeof(double));

	printf("We need about %d Mb in the GPU\n", sizetotal*sizeof(double)/1024/1024);

	/* To call CUDA */
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x, (dim+BLOCK_SIZE-1)/dimBlock.y);

	/* Inicializamos la malla de la función */
	cero<<<dimGrid, dimBlock>>>(f[m]);

	/* Initialize u[m] with random values */
	cero<<<dimGrid, dimBlock>>>(u[m]);
	random<<<dimGrid, dimBlock>>>(u[m]);

	/* main loop */
	for(i=0; i<N_MALLAS; i++){
		max_ant = max;
		max = 0.0;
		multigrid(&u[0],&f[0], &v[0], &d[0], m, &max, gpu_def, host_def);
		printf("[Iteration #%d] nd=%e ratio=%f\n", i, max, max/max_ant);
	}

	/* Free memory */
	for(i=0; i<N_MALLAS; i++){
		hipFree(u[i].v);
		hipFree(f[i].v);
		hipFree(v[i].v);
		hipFree(d[i].v);
	}
	hipFree(gpu_def);
	free(host_def);
	
	return 0;
}

void gpu_imprime(Grid g, const char *nombre){
	FILE *f;
	f=fopen(nombre, "w");
	int i,j;
	double *hg;
	hg = (double*)malloc(g.size*sizeof(double));
	hipMemcpy(hg,g.v, g.size*sizeof(double), hipMemcpyDeviceToHost);

	for(i=0; i< g.d; i++){
		for(j=0; j< g.d; j++){
			fprintf(f, "%d %d %f\n", i, j, hg[I(g.d,i,j)]);
		}
		fprintf(f,"\n");
	}
	fclose(f);
}

void gpu_muestra(Grid g, const char *nombre){
	int i,j;
	double *hg;
	hg = (double *)malloc(g.size*sizeof(double));
	hipMemcpy(hg, g.v, g.size*sizeof(double), hipMemcpyDeviceToHost);

	printf("%s=\n", nombre);
	for( i = 0 ; i<g.d; i++){
		for(j=0; j<g.d; j++){
			printf("%f ",hg[I(g.d,i,j)]);
		}
		printf("\n");
	}
}

void multigrid(Grid *u,
               Grid *f, 
							 Grid *v,
							 Grid *d,
							 int m,
							 double *max,
							 double *def,
							 double *host_def)
{
	int dim;
	int dim_;
	int i;
	
	/* Primer caso, malla 0, solución */
	if(m == 0){
		exacta<<<1,1>>>(u[m],f[m]);
	}
	else{ /* Some definitions to call cuda */
		dim  = (int)pow(2,m+1)+1;
		dim_ = (int)pow(2,m)+1;
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x, (dim+BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_((dim_+BLOCK_SIZE-1)/dimBlock.x,(dim_+BLOCK_SIZE-1)/dimBlock.y);

		/* some grids == 0 */
		cero<<<dimGrid , dimBlock>>>(v[m]);
		cero<<<dimGrid , dimBlock>>>(d[m]);
		cero<<<dimGrid_, dimBlock>>>(u[m-1]);
		cero<<<dimGrid_, dimBlock>>>(f[m-1]);

		/* R-N smoothing */
		suaviza_r<<<dimGrid, dimBlock>>>(u[m],f[m]);
		suaviza_n<<<dimGrid, dimBlock>>>(u[m],f[m]);

		/* Compute the defect */
		defecto<<<dimGrid, dimBlock>>>(u[m],f[m],d[m]);

		/* Defect from d[m] to f[m-1] */
		restringe<<<dimGrid_, dimBlock>>>(d[m],f[m-1]);

		/* Call to multigrid */
		multigrid(&u[0],&f[0],&v[0],&d[0],m-1,max, def, host_def);

		/* Interpolate from u[m-1] to v[m] */
		interpola<<<dimGrid_, dimBlock>>>(u[m-1], v[m]);

		/* Sum */
		suma<<<dimGrid, dimBlock>>>(u[m],v[m]);

		/* R-N smoothing */
		suaviza_r<<<dimGrid, dimBlock>>>(u[m],f[m]);
		suaviza_n<<<dimGrid, dimBlock>>>(u[m],f[m]);

		/* If we're in the upper grid, check the defect */
		if(m==N_MALLAS-1){
			defecto<<<dimGrid, dimBlock>>>(u[m],f[m],d[m]);
			dim3 dg((dim+BLOCK_SIZE-1)/dimBlock.x,1);
			dim3 db(BLOCK_SIZE,1);

			/* compute the max or each row */
			maxx<<<dg, db>>>(d[m],def);

			/* copy the vector to the host */
			hipMemcpy(host_def, def, dim*dim*sizeof(double), hipMemcpyDeviceToHost);

			max[0]=0.0;
			for(i=0;i<dim;i++){
				if(max[0]<host_def[i])
					max[0]=host_def[i];
			}
		}
	}
}
