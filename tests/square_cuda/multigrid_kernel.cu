
#include <hip/hip_runtime.h>
#define I(d,i,j) (i)*(d)+(j)

typedef struct{
	float *v;
	int d;
	int size;
} Grid;

__global__ void cero(Grid m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(1<=m.d && j<=m.d)
		m.v[I(m.d,i,j)]=0.0;
}


__global__ void random(Grid m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<m.d-1 && j< m.d-1 && i>0 && j>0) // Interior points
		m.v[I(m.d,i,j)]=10.1+sinf(i+cosf(j));
}

__global__ void suaviza_r(Grid u, Grid f){
	double h2 = pow(1.0/(u.d-1),2);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.d-1 && j<u.d-1 && i>0 && j>0){ // Interior points
		if((i+j)%2==0)
			u.v[I(u.d,i,j)]=0.25*(f.v[I(u.d,i  ,j  )]*h2
			                     +u.v[I(u.d,i-1,j  )]
													 +u.v[I(u.d,i+1,j  )]
													 +u.v[I(u.d,i  ,j-1)]
													 +u.v[I(u.d,i  ,j+1)]);
	}
}
__global__ void suaviza_n(Grid u, Grid f){
	double h2 = pow(1.0/(u.d-1),2);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.d-1 && j<u.d-1 && i>0 && j>0){ // Interior points
		if((i+j)%2==1)
			u.v[I(u.d,i,j)]=0.25*(f.v[I(u.d,i  ,j  )]*h2
			                     +u.v[I(u.d,i-1,j  )]
													 +u.v[I(u.d,i+1,j  )]
													 +u.v[I(u.d,i  ,j-1)]
													 +u.v[I(u.d,i  ,j+1)]);
	}
}

__global__ void defecto(Grid u, Grid f, Grid d){
	double h2 = pow(1.0/(u.d-1),2);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.d-1 && j<u.d-1 && i>0 && j > 0){ //Interior points
		d.v[I(u.d,i,j)] = f.v[I(u.d,i  ,j  )]
		              -(4*u.v[I(u.d,i  ,j  )]
									   -u.v[I(u.d,i-1,j  )]
										 -u.v[I(u.d,i+1,j  )]
										 -u.v[I(u.d,i  ,j-1)]
										 -u.v[I(u.d,i  ,j+1)])/h2;
	}
}

__global__ void restringe(Grid sup, Grid in){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<in.d-1 && j<in.d-1 && i>0 && j>0){ //Interior points
		in.v[I(in.d,i,j)] = (4* sup.v[I(sup.d,2*i  ,2*j  )]
		                    +2*(sup.v[I(sup.d,2*i-1,2*j  )]
												   +sup.v[I(sup.d,2*i+1,2*j  )]
													 +sup.v[I(sup.d,2*i  ,2*j-1)]
													 +sup.v[I(sup.d,2*i  ,2*j+1)])
													 +sup.v[I(sup.d,2*i-1,2*j-1)]
													 +sup.v[I(sup.d,2*i-1,2*j+1)]
													 +sup.v[I(sup.d,2*i+1,2*j-1)]
													 +sup.v[I(sup.d,2*i+1,2*j+2)])/16;
	}
}

__global__ void exacta(Grid u, Grid f){
	u.v[I(u.d,1,1)]=f.v[I(u.d,1,1)]/16;
}

__global__ void interpola(Grid u, Grid v){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.d && j<u.d){
		v.v[I(v.d,2*i,2*j)]         = u.v[I(u.d,i,j)];
		if(2*i+1<v.d)
			v.v[I(v.d,2*i+1,2*j  )]   = (u.v[I(u.d,i,j)]+u.v[I(u.d,i+1,j)])/2;
		if(2*j+1<v.d)
			v.v[I(v.d,2*i  ,2*j+1)]   = (u.v[I(u.d,i,j)]+u.v[I(u.d,i  ,j+1)])/2;
		if(2*i+1<v.d && 2*j+1<v.d) 
			v.v[I(v.d,2*i+1, 2*j+1)]  = (u.v[I(u.d,i,j)]+u.v[I(u.d,i+1,j  )]
																+ u.v[I(u.d,i, j+1)]+u.v[I(u.d, i+1, j+1)])/4;
	}
}

__global__ void suma(Grid u, Grid v){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.d && j<u.d){
		u.v[I(u.d,i,j)]+=v.v[I(u.d,i,j)];
	}
}

__global__ void maxx(Grid d, double *def)
{
	int i = blockIdx.x + blockDim.x + threadIdx.x;
	int j;
	def[i]=0.0;
	for(j=0;j<d.d;j++){
		if(abs(d.v[I(d.d,i,j)])>def[i]){
			def[i]=abs(d.v[I(d.d,i,j)]);
		}
	}
}


